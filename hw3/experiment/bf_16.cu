#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <algorithm>
#include <pthread.h>
#include <iostream>
#include <zlib.h>
#include <cstdlib>
#include <cassert>
using namespace std;

#define blocksize 16
// phase2 need 3 arr --> 49152 / 4(int) / 3 = 4096
// arr is square --> 4096 = 64 * 64 --> blocksize = 64
// (64 * 64) / 1024(number of threads per block) = 4 per thread
// each thread is responsible for 4 data
int* d;
int V, E;
int V_sq;
int original_V;
// If there is no valid path between i->j, dist(i, j) = 2 ^ 30 − 1 = 1073741823.
const int MAXIMUM = ((1 << 30) - 1);


void handle_input(char* input_file){
    FILE* file = fopen(input_file, "rb");

    // Read the number of vertices and edges
    fread(&V, sizeof(int), 1, file);
    fread(&E, sizeof(int), 1, file);
    // for blocksize = 64
    original_V = V;
    if(V % blocksize != 0){
        V += (blocksize - V % blocksize);
    }
    V_sq = V * V;
    d = (int*)malloc(V_sq * sizeof(int));
    int i;
    for(i = 0; i < V; i ++){
        for(int j = 0; j < V; j ++){
            int idx = i * V + j;
            if(i == j)d[idx] = 0;
            else d[idx] = MAXIMUM;
        }
    }

    for(i = 0; i < E; i ++){
        int src, dst, dist;
        fread(&src, sizeof(int), 1, file);
        fread(&dst, sizeof(int), 1, file);
        fread(&dist, sizeof(int), 1, file);
        d[src * V + dst] = dist;
    }

    fclose(file);
    return;
}

__global__ void Phase1(int* d, int round, int V){
    // put data to shared memory
    __shared__ int shared[blocksize][blocksize];
    int i = threadIdx.x;
    int j = threadIdx.y;
    // real index in d
    int idx_x = i + round * blocksize;
    int idx_y = j + round * blocksize;
    int idx_d = idx_y * V + idx_x;
    shared[j][i] = d[idx_d];
    shared[j + 8][i] = d[idx_d + V * 8];
    shared[j][i + 8] = d[idx_d + 8];
    shared[j + 8][i + 8] = d[idx_d + V * 8 + 8];
    __syncthreads();
    // calculation
    #pragma unroll
    for(int k = 0; k < blocksize; k ++){
        shared[j][i] = min(shared[j][i], shared[j][k] + shared[k][i]);
        shared[j + 8][i] = min(shared[j + 8][i], shared[j + 8][k] + shared[k][i]);
        shared[j][i + 8] = min(shared[j][i + 8], shared[j][k] + shared[k][i + 8]);
        shared[j + 8][i + 8] = min(shared[j + 8][i + 8], shared[j + 8][k] + shared[k][i + 8]);
        __syncthreads();
    }

    //write back
    d[idx_d] = shared[j][i];
    d[idx_d + V * 8] = shared[j + 8][i];
    d[idx_d + 8] = shared[j][i + 8];
    d[idx_d + V * 8 + 8] = shared[j + 8][i + 8];
}

__global__ void Phase2(int* d, int round, int V){
    if(round == blockIdx.y)return;
    // put data to shared memory
    __shared__ int pivot[blocksize][blocksize];
    __shared__ int row[blocksize][blocksize];
    __shared__ int col[blocksize][blocksize];
    int i = threadIdx.x;
    int j = threadIdx.y;
    // real index in d
    int idx_x = i + round * blocksize;
    int idx_y = j + round * blocksize;
    int idx_x_row = i + blockIdx.y * blocksize; //y is fixed, only x change
    int idx_y_col = j + blockIdx.y * blocksize; //x is fixed, only y change
    // pivot
    int idx_d = idx_y * V + idx_x;
    pivot[j][i] = d[idx_d];
    pivot[j + 8][i] = d[idx_d + V * 8];
    pivot[j][i + 8] = d[idx_d + 8];
    pivot[j + 8][i + 8] = d[idx_d + V * 8 + 8];
    //row --> y fixed
    int idx_row_d = idx_y * V + idx_x_row;
    row[j][i] = d[idx_row_d];
    row[j + 8][i] = d[idx_row_d + V * 8];
    row[j][i + 8] = d[idx_row_d + 8];
    row[j + 8][i + 8] = d[idx_row_d + V * 8 + 8];
    //col --> x fixed
    int idx_col_d = idx_y_col * V + idx_x;
    col[j][i] = d[idx_col_d];
    col[j + 8][i] = d[idx_col_d + V * 8];
    col[j][i + 8] = d[idx_col_d + 8];
    col[j + 8][i + 8] = d[idx_col_d + V * 8 + 8];
    __syncthreads();
    #pragma unroll
    for(int k = 0; k < blocksize; k ++){
        row[j][i] = min(row[j][i], pivot[j][k] + row[k][i]);
        row[j + 8][i] = min(row[j + 8][i], pivot[j + 8][k] + row[k][i]);
        row[j][i + 8] = min(row[j][i + 8], pivot[j][k] + row[k][i + 8]);
        row[j + 8][i + 8] = min(row[j + 8][i + 8], pivot[j + 8][k] + row[k][i + 8]);

        col[j][i] = min(col[j][i], col[j][k] + pivot[k][i]);
        col[j + 8][i] = min(col[j + 8][i], col[j + 8][k] + pivot[k][i]);
        col[j][i + 8] = min(col[j][i + 8], col[j][k] + pivot[k][i + 8]);
        col[j + 8][i + 8] = min(col[j + 8][i + 8], col[j + 8][k] + pivot[k][i + 8]);
        __syncthreads();
    }

    //row --> y fixed
    d[idx_row_d] = row[j][i];
    d[idx_row_d + V * 8] = row[j + 8][i];
    d[idx_row_d + 8] = row[j][i + 8];
    d[idx_row_d + V * 8 + 8] = row[j + 8][i + 8];
    //col --> x fixed
    d[idx_col_d] = col[j][i];
    d[idx_col_d + V * 8] = col[j + 8][i];
    d[idx_col_d + 8] = col[j][i + 8];
    d[idx_col_d + V * 8 + 8] = col[j + 8][i + 8];
}

__global__ void Phase3(int* d, int round, int V){
    // put data to shared memory
    if(round == blockIdx.x || round == blockIdx.y)return;
    __shared__ int pivot[blocksize][blocksize];
    __shared__ int row[blocksize][blocksize];
    __shared__ int col[blocksize][blocksize];
    int i = threadIdx.x;
    int j = threadIdx.y;
    // real index in d
    int idx_x = i + round * blocksize;
    int idx_y = j + round * blocksize;
    int idx_x_row = i + blockIdx.x * blocksize; //y is fixed, only x change
    int idx_y_col = j + blockIdx.y * blocksize; //x is fixed, only y change
    // pivot
    int idx_d = idx_y_col * V + idx_x_row;
    pivot[j][i] = d[idx_d];
    pivot[j + 8][i] = d[idx_d + V * 8];
    pivot[j][i + 8] = d[idx_d + 8];
    pivot[j + 8][i + 8] = d[idx_d + V * 8 + 8];
    //row --> y fixed
    int idx_row_d = idx_y_col * V + idx_x;
    row[j][i] = d[idx_row_d];
    row[j + 8][i] = d[idx_row_d + V * 8];
    row[j][i + 8] = d[idx_row_d + 8];
    row[j + 8][i + 8] = d[idx_row_d + V * 8 + 8];
    //col --> x fixed
    int idx_col_d = idx_y * V + idx_x_row;
    col[j][i] = d[idx_col_d];
    col[j + 8][i] = d[idx_col_d + V * 8];
    col[j][i + 8] = d[idx_col_d + 8];
    col[j + 8][i + 8] = d[idx_col_d + V * 8 + 8];
    __syncthreads();
    #pragma unroll
    for(int k = 0; k < blocksize; k ++){
        pivot[j][i] = min(pivot[j][i], row[j][k] + col[k][i]);
        pivot[j + 8][i] = min(pivot[j + 8][i], row[j + 8][k] + col[k][i]);
        pivot[j][i + 8] = min(pivot[j][i + 8], row[j][k] + col[k][i + 8]);
        pivot[j + 8][i + 8] = min(pivot[j + 8][i + 8], row[j + 8][k] + col[k][i + 8]);
    }

    d[idx_d] = pivot[j][i];
    d[idx_d + V * 8] = pivot[j + 8][i];
    d[idx_d + 8] = pivot[j][i + 8];
    d[idx_d + V * 8 + 8] = pivot[j + 8][i + 8];
}



void handle_output(char* output_file){
    FILE *file = fopen(output_file, "w");
    for(int i = 0; i < original_V; i ++){
        fwrite(d + i * V, sizeof(int), original_V, file);
    }
    fclose(file);
    return;
}
int main(int argc, char** argv) {

    handle_input(argv[1]);

    // for GPU code
    size_t d_size = V_sq * sizeof(int);// use size_t instead of int !!!
    hipHostRegister(d, d_size, hipHostRegisterDefault);
    int* device_d;
    hipMalloc(&device_d, d_size);
    hipMemcpy(device_d, d, d_size, hipMemcpyHostToDevice);
    dim3 blk(8, 8);
    int total_rnd = V / blocksize;
    dim3 grid_ph2(1, total_rnd);
    dim3 grid_ph3(total_rnd, total_rnd);
    for(int k = 0; k < total_rnd; k ++){
        //dot
        Phase1 << <1, blk>>> (device_d, k, V);
        //line
        Phase2 << <grid_ph2, blk>>>(device_d, k, V);
        //plane
        Phase3 << <grid_ph3, blk>>>(device_d, k, V);
    }

    hipMemcpy(d, device_d, d_size, hipMemcpyDeviceToHost);
    handle_output(argv[2]);
    return 0;
}
