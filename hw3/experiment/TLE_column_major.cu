#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <algorithm>
#include <pthread.h>
#include <iostream>
#include <zlib.h>
#include <cstdlib>
#include <cassert>
using namespace std;

#define blocksize 64
// phase2 need 3 arr --> 49152 / 4(int) / 3 = 4096
// arr is square --> 4096 = 64 * 64 --> blocksize = 64
// (64 * 64) / 1024(number of threads per block) = 4 per thread
// each thread is responsible for 4 data
int* d;
int V, E;
int V_sq;
int original_V;
// If there is no valid path between i->j, dist(i, j) = 2 ^ 30 − 1 = 1073741823.
const int MAXIMUM = ((1 << 30) - 1);


void handle_input(char* input_file){
    FILE* file = fopen(input_file, "rb");

    // Read the number of vertices and edges
    fread(&V, sizeof(int), 1, file);
    fread(&E, sizeof(int), 1, file);
    // for blocksize = 64
    original_V = V;
    if(V % blocksize != 0){
        V += (blocksize - V % blocksize);
    }
    V_sq = V * V;
    d = (int*)malloc(V_sq * sizeof(int));
    int i;
    for(i = 0; i < V; i ++){
        for(int j = 0; j < V; j ++){
            int idx = i * V + j;
            if(i == j)d[idx] = 0;
            else d[idx] = MAXIMUM;
        }
    }

    for(i = 0; i < E; i ++){
        int src, dst, dist;
        fread(&src, sizeof(int), 1, file);
        fread(&dst, sizeof(int), 1, file);
        fread(&dist, sizeof(int), 1, file);
        d[src * V + dst] = dist;
    }

    fclose(file);
    return;
}

__global__ void Phase1(int* d, int round, int V){
    // put data to shared memory
    __shared__ int shared[blocksize][blocksize];
    int i = threadIdx.x ;
    int j = threadIdx.y * 4;
    // printf("round = %d, i = %d, j = %d\n", round, i, j);
    // real index in d
    int idx_x = i + round * blocksize;
    int idx_y = j + round * blocksize;
    int idx_d = idx_y * V + idx_x;
    shared[j][i] = d[idx_d];
    shared[j + 1][i] = d[idx_d + V];
    shared[j + 2][i] = d[idx_d + 2 * V];
    shared[j + 3][i] = d[idx_d + 3 * V];
    // printf("d[%d][%d] = %d, d[%d][%d] = %d, d[%d][%d] = %d, d[%d][%d] = %d\n", idx_y, idx_x, shared[j][i], \
    //     idx_y+32, idx_x, shared[j + 32][i], idx_y, idx_x+32, shared[j][i + 32], idx_y+32, idx_x+32, shared[j + 32][i + 32]);
    __syncthreads();
    // calculation
    #pragma unroll
    for(int k = 0; k < blocksize; k ++){
        shared[j][i] = min(shared[j][i], shared[j][k] + shared[k][i]);
        shared[j + 1][i] = min(shared[j + 1][i], shared[j + 1][k] + shared[k][i]);
        shared[j + 2][i] = min(shared[j + 2][i], shared[j + 2][k] + shared[k][i]);
        shared[j + 3][i] = min(shared[j + 3][i], shared[j + 3][k] + shared[k][i]);
        __syncthreads();
    }

    //write back
    d[idx_d] = shared[j][i];
    d[idx_d + V] = shared[j + 1][i];
    d[idx_d + 2 * V] = shared[j + 2][i];
    d[idx_d + 3 * V] = shared[j + 3][i];
    // printf("d[%d][%d] = %d, d[%d][%d] = %d, d[%d][%d] = %d, d[%d][%d] = %d\n", idx_y, idx_x, d[idx_d], \
    //     idx_y+32, idx_x, d[idx_d + V * 32], idx_y, idx_x+32, d[idx_d + 32], idx_y+32, idx_x+32, d[idx_d + V * 32 + 32]);
}

__global__ void Phase2(int* d, int round, int V){
    if(round == blockIdx.y)return;
    // printf("blockIdx.x = %d, blockIdx.y = %d\n", blockIdx.x, blockIdx.y);
    // put data to shared memory
    __shared__ int pivot[blocksize][blocksize];
    __shared__ int row[blocksize][blocksize];
    __shared__ int col[blocksize][blocksize];
    int i = threadIdx.x;
    int j = threadIdx.y * 4;
    // real index in d
    int idx_x = i + round * blocksize;
    int idx_y = j + round * blocksize;
    int idx_x_row = i + blockIdx.y * blocksize; //y is fixed, only x change
    int idx_y_col = j + blockIdx.y * blocksize; //x is fixed, only y change
    // pivot
    int idx_d = idx_y * V + idx_x;
    pivot[j][i] = d[idx_d];
    pivot[j + 1][i] = d[idx_d + V];
    pivot[j + 2][i] = d[idx_d + 2 * V];
    pivot[j + 3][i] = d[idx_d + 3 * V];
    //row --> y fixed
    int idx_row_d = idx_y * V + idx_x_row;
    row[j][i] = d[idx_row_d];
    row[j + 1][i] = d[idx_row_d + V];
    row[j + 2][i] = d[idx_row_d + 2 * V];
    row[j + 3][i] = d[idx_row_d + 3 * V];
    //col --> x fixed
    int idx_col_d = idx_y_col * V + idx_x;
    col[j][i] = d[idx_col_d];
    col[j + 1][i] = d[idx_col_d + V];
    col[j + 2][i] = d[idx_col_d + 2 * V];
    col[j + 3][i] = d[idx_col_d + 3 * V];
    __syncthreads();
    #pragma unroll
    for(int k = 0; k < blocksize; k ++){
        row[j][i] = min(row[j][i], pivot[j][k] + row[k][i]);
        row[j + 1][i] = min(row[j + 1][i], pivot[j + 1][k] + row[k][i]);
        row[j + 2][i] = min(row[j + 2][i], pivot[j + 2][k] + row[k][i]);
        row[j + 3][i] = min(row[j + 3][i], pivot[j + 3][k] + row[k][i]);

        col[j][i] = min(col[j][i], col[j][k] + pivot[k][i]);
        col[j + 1][i] = min(col[j + 1][i], col[j + 1][k] + pivot[k][i]);
        col[j + 2][i] = min(col[j + 2][i], col[j + 2][k] + pivot[k][i]);
        col[j + 3][i] = min(col[j + 3][i], col[j + 3][k] + pivot[k][i]);
        __syncthreads();
    }

    //row --> y fixed
    d[idx_row_d] = row[j][i];
    d[idx_row_d + V] = row[j + 1][i];
    d[idx_row_d + 2 * V] = row[j + 2][i];
    d[idx_row_d + 3 * V] = row[j + 3][i];
    //col --> x fixed
    d[idx_col_d] = col[j][i];
    d[idx_col_d + V] = col[j + 1][i];
    d[idx_col_d + 2 * V] = col[j + 2][i];
    d[idx_col_d + 3 * V] = col[j + 3][i];
}

__global__ void Phase3(int* d, int round, int V){
    // put data to shared memory
    // printf("blockIdx.x = %d, blockIdx.y = %d\n", blockIdx.x, blockIdx.y);
    if(round == blockIdx.x || round == blockIdx.y)return;
    __shared__ int pivot[blocksize][blocksize];
    __shared__ int row[blocksize][blocksize];
    __shared__ int col[blocksize][blocksize];
    int i = threadIdx.x;
    int j = threadIdx.y * 4;
    // real index in d
    int idx_x = i + round * blocksize;
    int idx_y = j + round * blocksize;
    int idx_x_row = i + blockIdx.x * blocksize; //y is fixed, only x change
    int idx_y_col = j + blockIdx.y * blocksize; //x is fixed, only y change
    // pivot
    int idx_d = idx_y_col * V + idx_x_row;
    pivot[j][i] = d[idx_d];
    pivot[j + 1][i] = d[idx_d + V];
    pivot[j + 2][i] = d[idx_d + 2 * V];
    pivot[j + 3][i] = d[idx_d + 3 * V];
    //row --> y fixed
    int idx_row_d = idx_y_col * V + idx_x;
    row[j][i] = d[idx_row_d];
    row[j + 1][i] = d[idx_row_d + V];
    row[j + 2][i] = d[idx_row_d + 2 * V];
    row[j + 3][i] = d[idx_row_d + 3 * V];
    //col --> x fixed
    int idx_col_d = idx_y * V + idx_x_row;
    col[j][i] = d[idx_col_d];
    col[j + 1][i] = d[idx_col_d + V];
    col[j + 2][i] = d[idx_col_d + 2 * V];
    col[j + 3][i] = d[idx_col_d + 3 * V];
    __syncthreads();
    #pragma unroll
    for(int k = 0; k < blocksize; k ++){
        pivot[j][i] = min(pivot[j][i], row[j][k] + col[k][i]);
        pivot[j + 1][i] = min(pivot[j + 1][i], row[j + 1][k] + col[k][i]);
        pivot[j + 2][i] = min(pivot[j + 2][i], row[j + 2][k] + col[k][i]);
        pivot[j + 3][i] = min(pivot[j + 3][i], row[j + 3][k] + col[k][i]);
    }

    d[idx_d] = pivot[j][i];
    d[idx_d + V] = pivot[j + 1][i];
    d[idx_d + 2 * V] = pivot[j + 2][i];
    d[idx_d + 3 * V] = pivot[j + 3][i];
}



void handle_output(char* output_file){
    FILE *file = fopen(output_file, "w");
    for(int i = 0; i < original_V; i ++){
        fwrite(d + i * V, sizeof(int), original_V, file);
    }
    fclose(file);
    return;
}
int main(int argc, char** argv) {

    handle_input(argv[1]);

    // for GPU code
    size_t d_size = V_sq * sizeof(int);// use size_t instead of int !!!
    hipHostRegister(d, d_size, hipHostRegisterDefault);
    int* device_d;
    hipMalloc(&device_d, d_size);
    hipMemcpy(device_d, d, d_size, hipMemcpyHostToDevice);
    dim3 blk(64, 16);
    int total_rnd = V / blocksize;
    dim3 grid_ph2(1, total_rnd);
    dim3 grid_ph3(total_rnd, total_rnd);
    for(int k = 0; k < total_rnd; k ++){
        //dot
        Phase1 << <1, blk>>> (device_d, k, V);
        //line
        Phase2 << <grid_ph2, blk>>>(device_d, k, V);
        //plane
        Phase3 << <grid_ph3, blk>>>(device_d, k, V);
    }

    hipMemcpy(d, device_d, d_size, hipMemcpyDeviceToHost);
    handle_output(argv[2]);
    // free(d);
    return 0;
}
